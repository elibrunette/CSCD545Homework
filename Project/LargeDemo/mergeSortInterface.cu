#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "ArrayUtils.h"
#include "./MergeSort/merge_sort.cu"
#include "./MergeSort/mergeSort_common.h"

/*
double * mergeHelper(double * arr, int n);

double ** mergeSort(double ** arr, int classCol, int classRow, int testCol, int testRow) {
	int n = classRow;
	int x = 0; 
	for(x = 0; x < classRow; x++) {
		arr[x] = mergeHelper(arr[x], n);
	}
	return arr;
}

double * mergeHelper(double * arr, int n) {
	double * toReturn = (double *) calloc(n, sizeof(double));
	int i = 0; 
	int j = 0;
	int smallest;
	double temp;

	for(i = 0; i < n; i++) {
		smallest = i;
		for(j = 0; j < n; j++) {
			if(arr[j] < arr[smallest]) 
				smallest = j;
		}
		temp = arr[i];
		arr[i] = arr[smallest];
		arr[smallest] = temp;
	}

	free(arr);
	return toReturn;
}
*/

void fillValues(int * arr, int N) {
	uint x = 0; 
	for(x = 0; x < N; x++) {
		arr[x] = x;
	} 
}

int ** mergeSortForKNN(double ** arr, int classCol, int classRow, int testCol, int testRow) {

	int *h_SrcVal = (int *) malloc(testRow * testCol * sizeof(int)); //h_label
	int ** toReturn = createInt2DArray(classRow, testRow);

    	double  *d_SrcKey, *d_BufKey, *d_DstKey, *h_dist_temp;
    	int     *d_SrcVal, *d_BufVal, *d_DstVal, *h_label_temp;
	//print2DDoubleArray(arr, classRow, testRow);
	//printf("Incoming perameters classCol: %d\nclassRow: %d\ntestCol: %d\ntestRow:%d\n", classCol, classRow, testCol, testRow);

	//setting up variables for sorting 
	int N = pow(2, ceil(log(classRow)/log(2)));

	if(N < SHARED_SIZE_LIMIT)
		N = SHARED_SIZE_LIMIT;

	fillValues(h_SrcVal, N);
	hipMalloc((void **)&d_SrcKey, N * sizeof(double));
	hipMalloc((void **)&d_DstKey, N * sizeof(double));
	hipMalloc((void **)&d_BufKey, N * sizeof(double));
	hipMalloc((void **)&d_SrcVal, N * sizeof(int));
	hipMalloc((void **)&d_DstVal, N * sizeof(int));
	hipMalloc((void **)&d_BufVal, N * sizeof(int));

	//printSingleIntArray(h_SrcVal, N);


	
	initMergeSort();

	int i = 0;
	int j = 0;

	h_dist_temp = (double *) malloc(N * sizeof(double));
	h_label_temp = (int *) malloc(N * sizeof(int));

	//printf("Made up to the double for loop for sorting the array\n");
	for(i = 0; i < testRow; i++) { 		//iterates through the arr array and creates a temp array that gets padded
			
		//allocating memory on GPU 

		//printf("Filled Values\n");



		//printf("i: %d\n", i);
		for(j = 0; j < N; j++) {
			if(j < classRow) {
				//printf("arr[i][j]: %f\n" , arr[i][j]);
				h_dist_temp[j] = arr[i][j];
				h_label_temp[j] = h_SrcVal[j];
			}
			else {
				h_dist_temp[j] = INFINITY; //10000 is out of range of the rng used for this program
				h_label_temp[j] = 0;
			}
		}
		//printSingleArray(h_dist_temp, N);

		hipMemcpy(d_SrcKey, h_dist_temp, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_SrcVal, h_label_temp, N * sizeof(int), hipMemcpyHostToDevice);
		
		//printf("Made it to right before GPU MergeSort\n");
		//printf("Value for N: %d\n", N);

        		mergeSort(d_DstKey, d_DstVal, d_BufKey, d_BufVal, d_SrcKey, d_SrcVal, N, 1);
		//printf("Right after the mergeSort\n");
		
		hipMemcpy(arr[i], d_DstKey, classRow * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(toReturn[i], d_DstVal, classRow * sizeof(int), hipMemcpyDeviceToHost);

		//printf("copying back to the result arr\n");
		//copy computed values back to original array
	/*		
		int x = 0;
		for(x = 0; x < classRow; x++) {
			arr[i][x] = h_dist_temp[x];
			toReturn[i][x] = h_SrcVal[x];
		}
	*/
	}

	//printf("made it to right after the for loop\n");
	closeMergeSort();

    	hipFree(d_SrcKey);
    	hipFree(d_DstKey);
    	hipFree(d_BufKey);
    	hipFree(d_DstVal);
    	hipFree(d_SrcVal);
    	hipFree(d_BufVal);

    	free(h_dist_temp);
    	free(h_label_temp);

	//printf("Made it out of merge and right before the print arr command");
	//print2DDoubleArray(arr, classRow, testRow);
	//print2DArray(toReturn, classRow, testRow);
	return toReturn;
}











